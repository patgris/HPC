#include <stdio.h>
#include <time.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>


#define V 24
#define E 36
#define MAX_WEIGHT 1000000
#define TRUE    1
#define FALSE   0


typedef int boolean;
//
//Represents an edge or path between Vertices
typedef struct
{
	int u;
	int v;

} Edge;

//Represents a Vertex
typedef struct 
{
	int title;
	boolean visited;	

} Vertex;

__constant__ int M[E];
__constant__ Edge M2[E];

const Edge edd[E] = {{0, 4}, {0, 6}, {0,2}, {4,6}, {4,7}, {0, 7}, {7, 3}, {3, 1}, {2,5}, {2, 1}, {5,3}, {2,6}, {6,9}, {5,12}, {8,10}, {12,15}, {10,12}, {7, 9}, {7, 10}, {8, 10}, {7,8 },{8,9 },{9,10 },{10,11 },{11,12 },{12,13 },{13,14 },{14,15 },{15,16 },{16,17 },{17,18 },{18,19},{19,20 },{20,21},{21,22 },{22, 23}};
const int ww[E] = {10, 90, 30, 20, 20, 50, 10, 20, 10, 10, 10, 70, 10, 90, 30, 20, 20, 50, 10, 20, 10, 10, 10, 20, 10, 90, 30, 20, 20, 50, 10, 20, 10, 10, 10, 20};

//Finds the weight of the path from vertex u to vertex v
__device__ int findEdgeD(Vertex u, Vertex v)
{
	
	int i;
	for(i = 0; i < E; i++)
	{

		if(M2[i].u == u.title && M2[i].v == v.title)
		{
			return M[i];
		}
	}

	return MAX_WEIGHT;

}

int findEdge(Vertex u, Vertex v)
{
	
	int i;
	for(i = 0; i < E; i++)
	{

		if(edd[i].u == u.title && edd[i].v == v.title)
		{
			return ww[i];
		}
	}

	return MAX_WEIGHT;

}

//Finds the branches of the vertex
__global__ void Find_Vertex(Vertex *vertices, int *length, int *updateLength)
{

	int u = threadIdx.x;


	if(vertices[u].visited == FALSE)
	{
		

		vertices[u].visited = TRUE;


		int v;

		for(v = 0; v < V; v++)
		{	
			//Find the weight of the edge
			int weight = findEdgeD(vertices[u], vertices[v]);

			//Checks if the weight is a candidate
			if(weight < MAX_WEIGHT)
			{	
				//If the weight is shorter than the current weight, replace it
				if(updateLength[v] > length[u] + weight)
				{
					updateLength[v] = length[u] + weight;
				}
			}
		}

	}
	
}

//Updates the shortest path array (length)
__global__ void Update_Paths(Vertex *vertices, int *length, int *updateLength)
{
	int u = threadIdx.x;
	if(length[u] > updateLength[u])
	{

		length[u] = updateLength[u];
		vertices[u].visited = FALSE;
	}

	updateLength[u] = length[u];


}


//Prints the an array of elements
void printArray(int *array)
{
	int i;
	for(i = 0; i < V; i++)
	{
		printf("Shortest Path to Vertex: %d is %d\n", i, array[i]);
	}
}


//Runs the program
int main(void)
{

	//Variables for the Host Device
	Vertex *vertices;

	//Len is the shortest path and updateLength is a special array for modifying updates to the shortest path
	int *len, *updateLength;
	


	//Pointers for the CUDA device
	Vertex *d_V;
	Edge *d_E;
	int *d_L;
	int *d_C;
  
  int sizeM = sizeof(int)*E; 

	//Sizes used for allocation
	int sizeV = sizeof(Vertex) * V;
	int sizeE = sizeof(Edge) * E;
	int size = V * sizeof(int);


	//Timer initialization
	float runningTime;
	hipEvent_t timeStart, timeEnd;


	//Creates the timers
	hipEventCreate(&timeStart);
	hipEventCreate(&timeEnd);


	//Allocates space for the variables
	vertices = (Vertex *)malloc(sizeV);
	len = (int *)malloc(size);
	updateLength = (int *)malloc(size);

	
	//----------------------------------Graph Base Test-------------------------------------//
	Edge ed[E] = {{0, 4}, {0, 6}, {0,2}, {4,6}, {4,7}, {0, 7}, {7, 3}, {3, 1}, {2,5}, {2, 1}, {5,3}, {2,6}, {6,9}, {5,12}, {8,10}, {12,15}, {10,12}, {7, 9}, {7, 10}, {8, 10}, {7,8 },{8,9 },{9,10 },{10,11 },{11,12 },{12,13 },{13,14 },{14,15 },{15,16 },{16,17 },{17,18 },{18,19},{19,20 },{20,21},{21,22 },{22, 23}};
	int w[E] = {10, 90, 30, 20, 20, 50, 10, 20, 10, 10, 10, 70, 10, 90, 30, 20, 20, 50, 10, 20, 10, 10, 10, 20, 10, 90, 30, 20, 20, 50, 10, 20, 10, 10, 10, 20};

  
	int i = 0;
	for(i = 0; i < V; i++)
	{
		Vertex a = { .title =i , .visited=FALSE};
		vertices[i] = a;


	}

	//----------------------------------Graph Base Test-------------------------------------//


	//--------------------------------Graph Randomizer-----------------------------------//
	// srand(time(NULL));
	// int i = 0;
	// for(i = 0; i < V; i++)
	// {
	// 	Vertex a = { .title =(int) i, .visited=FALSE};
	// 	vertices[i] = a;


	// }



	// for(i = 0; i < E; i++)
	// {

	// 	Edge e = {.u = (int) rand()%V , .v = rand()%V};
	// 	edges[i] = e;

	// 	weights[i] = rand()%100;

	// }

	//--------------------------------Graph Randomizer-----------------------------------//


	//Allocate space on the device
	hipMalloc((void**)&d_V, sizeV);
	hipMalloc((void**)&d_E, sizeE);
	hipMalloc((void**)&d_L, size);
	hipMalloc((void**)&d_C, size);

	//Initial Node
	Vertex root = {0, FALSE};


	//--------------------------------------Dijkstra's Algorithm--------------------------------------//
	root.visited = TRUE;
	
	
	len[root.title] = 0;
	updateLength[root.title] = 0;

	//Copy variables to the Device
	hipMemcpy(d_V, vertices, sizeV, hipMemcpyHostToDevice);
	hipMemcpy(d_L, len, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, updateLength, size, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(M),w,sizeM);
  	hipMemcpyToSymbol(HIP_SYMBOL(M2),ed,sizeE);
	

	//Loop finds the initial paths from the node 's'
	for(i = 0; i < V;i++)
	{

		if(vertices[i].title != root.title)
		{
			len[(int)vertices[i].title] = findEdge(root, vertices[i]);
			updateLength[vertices[i].title] = len[(int)vertices[i].title];
			
			

		}
		else{
		
			vertices[i].visited = TRUE;
		}


	}

	//Start the timer
	hipEventRecord(timeStart, 0);
		
	//Recopy the variables	
	hipMemcpy(d_L, len, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, updateLength, size, hipMemcpyHostToDevice);
					
	//Parallelization
	for(i = 0; i < V; i++){

	//Find_Vertex<<<1, V>>>(d_V, d_E, d_W, d_L, d_C);
    	Find_Vertex<<<1, V>>>(d_V, d_L, d_C);

	Update_Paths<<<1,V>>>(d_V, d_L, d_C);
			
	}	
	
	//Timing Events
	hipEventRecord(timeEnd, 0);
	hipEventSynchronize(timeEnd);
	hipEventElapsedTime(&runningTime, timeStart, timeEnd);

	//Copies the results back
	hipMemcpy(len, d_L, size, hipMemcpyDeviceToHost);

	printArray(len);

	//Running Time
	printf("Running Time: %f ms\n", runningTime);

	//--------------------------------------Dijkstra's Algorithm--------------------------------------//

	//Free up the space
	free(vertices);
	free(len);
	free(updateLength);
	hipFree(d_V);
	hipFree(d_L);
	hipFree(d_C);
	hipEventDestroy(timeStart);
	hipEventDestroy(timeEnd);

}
