#include <stdio.h>
#include <time.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>


#define V 24
#define E 36
#define MAX_WEIGHT 1000000
#define TRUE    1
#define FALSE   0

typedef int boolean;
//
//Represents an edge or path between Vertices
typedef struct
{
	int u;
	int v;

} Edge;

//Represents a Vertex
typedef struct 
{
	int title;
	boolean visited;	

} Vertex;


//Finds the weight of the path from vertex u to vertex v
__device__ __host__ int findEdge(Vertex u, Vertex v, Edge *edges, int *weights)
{

	int i;
	for(i = 0; i < E; i++)
	{

		if(edges[i].u == u.title && edges[i].v == v.title)
		{
			return weights[i];
		}
	}

	return MAX_WEIGHT;

}

//Finds the branches of the vertex
__global__ void Find_Vertex(Vertex *vertices, Edge *edges, int *weights, int *length, int *updateLength)
{

	int u = threadIdx.x;


	if(vertices[u].visited == FALSE)
	{
		

		vertices[u].visited = TRUE;


		int v;

		for(v = 0; v < V; v++)
		{	
			//Find the weight of the edge
			int weight = findEdge(vertices[u], vertices[v], edges, weights);

			//Checks if the weight is a candidate
			if(weight < MAX_WEIGHT)
			{	
				//If the weight is shorter than the current weight, replace it
				if(updateLength[v] > length[u] + weight)
				{
					updateLength[v] = length[u] + weight;
				}
			}
		}

	}
	
}

//Updates the shortest path array (length)
__global__ void Update_Paths(Vertex *vertices, int *length, int *updateLength)
{
	int u = threadIdx.x;
	if(length[u] > updateLength[u])
	{

		length[u] = updateLength[u];
		vertices[u].visited = FALSE;
	}

	updateLength[u] = length[u];


}


//Prints the an array of elements
void printArray(int *array)
{
	int i;
	for(i = 0; i < V; i++)
	{
		printf("Shortest Path to Vertex: %d is %d\n", i, array[i]);
	}
}


//Runs the program
int main(void)
{

	//Variables for the Host Device
	Vertex *vertices;	
	Edge *edges;

	//Weight of the paths
	int *weights;

	//Len is the shortest path and updateLength is a special array for modifying updates to the shortest path
	int *len, *updateLength;
	


	//Pointers for the CUDA device
	Vertex *d_V;
	Edge *d_E;
	int *d_W;
	int *d_L;
	int *d_C;


	//Sizes used for allocation
	int sizeV = sizeof(Vertex) * V;
	int sizeE = sizeof(Edge) * E;
	int size = V * sizeof(int);


	//Timer initialization
	float runningTime;
	hipEvent_t timeStart, timeEnd;


	//Creates the timers
	hipEventCreate(&timeStart);
	hipEventCreate(&timeEnd);


	//Allocates space for the variables
	vertices = (Vertex *)malloc(sizeV);
	edges = (Edge *)malloc(sizeE);
	weights = (int *)malloc(E* sizeof(int));
	len = (int *)malloc(size);
	updateLength = (int *)malloc(size);

	
	//----------------------------------Graph Base Test-------------------------------------//
	Edge ed[E] = {{0, 4}, {0, 6}, {0,2}, {4,6}, {4,7}, {0, 7}, {7, 3}, {3, 1}, {2,5}, {2, 1}, {5,3}, {2,6}, {6,9}, {5,12}, {8,10}, {12,15}, {10,12}, {7, 9}, {7, 10}, {8, 10}, {7,8 },{8,9 },{9,10 },{10,11 },{11,12 },{12,13 },{13,14 },{14,15 },{15,16 },{16,17 },{17,18 },{18,19},{19,20 },{20,21},{21,22 },{22, 23}};
	int w[E] = {10, 90, 30, 20, 20, 50, 10, 20, 10, 10, 10, 70, 10, 90, 30, 20, 20, 50, 10, 20, 10, 10, 10, 20, 10, 90, 30, 20, 20, 50, 10, 20, 10, 10, 10, 20};

	int i = 0;
	for(i = 0; i < V; i++)
	{
		Vertex a = { .title =i , .visited=FALSE};
		vertices[i] = a;


	}

	for(i = 0; i < E; i++)
	{
		edges[i] = ed[i];
		weights[i] = w[i];
	}

	//----------------------------------Graph Base Test-------------------------------------//


	//--------------------------------Graph Randomizer-----------------------------------//
	// srand(time(NULL));
	// int i = 0;
	// for(i = 0; i < V; i++)
	// {
	// 	Vertex a = { .title =(int) i, .visited=FALSE};
	// 	vertices[i] = a;


	// }



	// for(i = 0; i < E; i++)
	// {

	// 	Edge e = {.u = (int) rand()%V , .v = rand()%V};
	// 	edges[i] = e;

	// 	weights[i] = rand()%100;

	// }

	//--------------------------------Graph Randomizer-----------------------------------//


	//Allocate space on the device
	hipMalloc((void**)&d_V, sizeV);
	hipMalloc((void**)&d_E, sizeE);
	hipMalloc((void**)&d_W, E * sizeof(int));
	hipMalloc((void**)&d_L, size);
	hipMalloc((void**)&d_C, size);



	//Initial Node
	Vertex root = {0, FALSE};
	

	




	//--------------------------------------Dijkstra's Algorithm--------------------------------------//
	root.visited = TRUE;
	
	
	len[root.title] = 0;
	updateLength[root.title] = 0;

	//Copy variables to the Device
	hipMemcpy(d_V, vertices, sizeV, hipMemcpyHostToDevice);
	hipMemcpy(d_E, edges, sizeE, hipMemcpyHostToDevice);
	hipMemcpy(d_W, weights, E * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_L, len, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, updateLength, size, hipMemcpyHostToDevice);

	

	int j;
	

	//Loop finds the initial paths from the node 's'
	for(i = 0; i < V;i++)
	{

		if(vertices[i].title != root.title)
		{
			len[(int)vertices[i].title] = findEdge(root, vertices[i], edges, weights);
			updateLength[vertices[i].title] = len[(int)vertices[i].title];
			
			

		}
		else{
		
			vertices[i].visited = TRUE;
		}


	}

	//Start the timer
	hipEventRecord(timeStart, 0);
		
	//Recopy the variables	
	hipMemcpy(d_L, len, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, updateLength, size, hipMemcpyHostToDevice);
					
	//Parallelization
	for(i = 0; i < V; i++){

			Find_Vertex<<<1, V>>>(d_V, d_E, d_W, d_L, d_C);

			for(j = 0; j < V; j++)
			{
				Update_Paths<<<1,V>>>(d_V, d_L, d_C);
			}
	}	
	
	//Timing Events
	hipEventRecord(timeEnd, 0);
	hipEventSynchronize(timeEnd);
	hipEventElapsedTime(&runningTime, timeStart, timeEnd);

	//Copies the results back
	hipMemcpy(len, d_L, size, hipMemcpyDeviceToHost);

	printArray(len);

	//Running Time
	printf("Running Time: %f ms\n", runningTime);

	//--------------------------------------Dijkstra's Algorithm--------------------------------------//

	


	


	

	//Free up the space
	free(vertices);
	free(edges);
	free(weights);
	free(len);
	free(updateLength);
	hipFree(d_V);
	hipFree(d_E);
	hipFree(d_W);
	hipFree(d_L);
	hipFree(d_C);
	hipEventDestroy(timeStart);
	hipEventDestroy(timeEnd);



}
